#include "hip/hip_runtime.h"
/*

   Copyright (c) 2006-2010, The Scripps Research Institute

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

   Author: Dr. Oleg Trott <ot14@columbia.edu>,
           The Olson Lab,
           The Scripps Research Institute

*/


#include "kernel.h"
#include "math.h"
#include <vector>
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime.h"
/* Original Include files */
#include "monte_carlo.h"
#include "coords.h"
#include "mutate.h"
#include "quasi_newton.h"
#include "model.h"
#include "precalculate.h"
#include "cache.h"
#include "ad4cache.h"

/* Below based on mutate_conf.cpp */

 __device__ __forceinline__ void quaternion_increment(float* q, const float* rotation, float epsilon_fl);

 __device__ __forceinline__ void normalize_angle(float* x);

 __device__ __forceinline__
void output_type_cuda_init(output_type_cuda_t* out, const float* ptr) {
	memcpy(out, ptr, sizeof(float) * (3 + 4 + MAX_NUM_OF_LIG_TORSION + MAX_NUM_OF_FLEX_TORSION));
	out->lig_torsion_size = ptr[3 + 4 + MAX_NUM_OF_LIG_TORSION + MAX_NUM_OF_FLEX_TORSION];
	// did not assign coords and e
}

 __device__ __forceinline__
void output_type_cuda_init_with_output(output_type_cuda_t* out_new, const output_type_cuda_t* out_old) {
	memcpy(out_new, out_old, sizeof(float) * (3 + 4 + MAX_NUM_OF_LIG_TORSION + MAX_NUM_OF_FLEX_TORSION));
	out_new->lig_torsion_size = out_old->lig_torsion_size;
	// assign e but not coords
	out_new->e = out_old->e;
}

 __device__ __forceinline__
void output_type_cuda_increment(output_type_cuda_t* x, const change_cuda_t* c, float factor, float epsilon_fl) {
	// position increment
	for (int k = 0; k < 3; k++) x->position[k] += factor * c->position[k];
	// orientation increment
	float rotation[3];
	for (int k = 0; k < 3; k++) rotation[k] = factor * c->orientation[k];
	quaternion_increment(x->orientation, rotation, epsilon_fl);

	// torsion increment
	for (int k = 0; k < x->lig_torsion_size; k++) {
		float tmp = factor * c->lig_torsion[k];
		normalize_angle(&tmp);
		x->lig_torsion[k] += tmp;
		normalize_angle(&(x->lig_torsion[k]));
	}
}

 __device__ __forceinline__
float norm3(const float* a) {
	return sqrt(pow(a[0], 2) + pow(a[1], 2) + pow(a[2], 2));
}

 __device__ __forceinline__
void random_inside_sphere_gpu(float *random_inside_sphere, hiprandStatePhilox4_32_10_t* state){
	float4 random_inside_sphere_fl;
	while(true) { // on average, this will have to be run about twice
		random_inside_sphere_fl = hiprand_uniform4(state); // ~ U[0,1]
		random_inside_sphere[0] = (random_inside_sphere_fl.x - 0.5)*2.0;
		random_inside_sphere[1] = (random_inside_sphere_fl.y - 0.5)*2.0;
		random_inside_sphere[2] = (random_inside_sphere_fl.z - 0.5)*2.0;
		random_inside_sphere[3] = random_inside_sphere_fl.w;
		float r = norm3(random_inside_sphere);
		if (r < 1){
			return;
		}
	}
}


 __device__ __forceinline__
void normalize_angle(float* x) {
	while (1) {
		if (*x >= -(M_PI) && *x <= (M_PI)) {
			break;
		}
		else if (*x > 3 * (M_PI)) {
			float n = (*x - (M_PI)) / (2 * (M_PI));
			*x -= 2 * (M_PI) * ceil(n);
		}
		else if (*x < 3 * -(M_PI)) {
			float n = (-*x - (M_PI)) / (2 * (M_PI));
			*x += 2 * (M_PI) * ceil(n);
		}
		else if (*x > (M_PI)) {
			*x -= 2 * (M_PI);
		}
		else if (*x < -(M_PI)) {
			*x += 2 * (M_PI);
		}
		else {
			break;
		}
	}
}

 __device__ __forceinline__
bool quaternion_is_normalized(float* q) {
	float q_pow = pow(q[0], 2) + pow(q[1], 2) + pow(q[2], 2) + pow(q[3], 2);
	float sqrt_q_pow = sqrt(q_pow);
	return (q_pow - 1 < 0.001) && (sqrt_q_pow - 1 < 0.001);
}

 __device__ __forceinline__
void angle_to_quaternion(float* q, const float* rotation, float epsilon_fl) {
	float angle = norm3(rotation);
	if (angle > epsilon_fl) {
		float axis[3] = { rotation[0] / angle, rotation[1] / angle ,rotation[2] / angle };
		normalize_angle(&angle);
		float c = cos(angle / 2);
		float s = sin(angle / 2);
		q[0] = c; q[1] = s * axis[0]; q[2] = s * axis[1]; q[3] = s * axis[2];
		return;
	}
	q[0] = 1; q[1] = 0; q[2] = 0; q[3] = 0;
	return;
}

// quaternion multiplication
 __device__ __forceinline__
void angle_to_quaternion_multi(float* qa, const float* qb) {
	float tmp[4] = { qa[0],qa[1],qa[2],qa[3] };
	qa[0] = tmp[0] * qb[0] - tmp[1] * qb[1] - tmp[2] * qb[2] - tmp[3] * qb[3];
	qa[1] = tmp[0] * qb[1] + tmp[1] * qb[0] + tmp[2] * qb[3] - tmp[3] * qb[2];
	qa[2] = tmp[0] * qb[2] - tmp[1] * qb[3] + tmp[2] * qb[0] + tmp[3] * qb[1];
	qa[3] = tmp[0] * qb[3] + tmp[1] * qb[2] - tmp[2] * qb[1] + tmp[3] * qb[0];
}

 __device__ __forceinline__
void quaternion_normalize_approx(float* q, float epsilon_fl) {
	const float s = pow(q[0], 2) + pow(q[1], 2) + pow(q[2], 2) + pow(q[3], 2);
	// Omit one assert()
	if (fabs(s - 1) < TOLERANCE)
		;
	else {
		const float a = sqrt(s);
		for (int i = 0; i < 4; i++) q[i] /= a;
	}
}

 __device__ __forceinline__
void quaternion_increment(float* q, const float* rotation, float epsilon_fl) {
	float q_old[4] = { q[0],q[1],q[2],q[3] };
	angle_to_quaternion(q, rotation, epsilon_fl);
	angle_to_quaternion_multi(q, q_old);
	quaternion_normalize_approx(q, epsilon_fl);
	// assert(quaternion_is_normalized(q)); // unnecessary
}


 __device__ __forceinline__
float vec_distance_sqr(float* a, float* b) {
	return pow(a[0] - b[0], 2) + pow(a[1] - b[1], 2) + pow(a[2] - b[2], 2);
}

 __device__ __forceinline__
float gyration_radius(				int				m_lig_begin,
									int				m_lig_end,
						const		atom_cuda_t*		atoms,
						const		m_coords_cuda_t*	m_coords_gpu,
						const		float*			m_lig_node_origin
) {
	float acc = 0;
	int counter = 0;
	float origin[3] = { m_lig_node_origin[0], m_lig_node_origin[1], m_lig_node_origin[2] };
	for (int i = m_lig_begin; i < m_lig_end; i++) {
		float current_coords[3] = { m_coords_gpu->coords[i][0], m_coords_gpu->coords[i][1], m_coords_gpu->coords[i][2] };
		if (atoms[i].types[0] != EL_TYPE_H) { // for el, we use the first element (atoms[i].types[0])
			acc += vec_distance_sqr(current_coords, origin);
			++counter;
		}
	}
	return (counter > 0) ? sqrt(acc / counter) : 0;
}

 __device__ __forceinline__
void mutate_conf_cuda(const	int	num_steps, output_type_cuda_t *c,
			hiprandStatePhilox4_32_10_t* state, 
			const int	m_lig_begin,
			const int	m_lig_end,
			const atom_cuda_t* atoms,
			const m_coords_cuda_t* m_coords_gpu,
			const float*		m_lig_node_origin_gpu,
			const float			epsilon_fl,
			const float			amplitude
) {
	int flex_torsion_size = 0; // FIX? 20210727
	int count_mutable_entities = 2 + c->lig_torsion_size + flex_torsion_size;
	int which = hiprand(state) % count_mutable_entities;

	float random_inside_sphere[4];
	random_inside_sphere_gpu(random_inside_sphere, state);
	if (which == 0){
		DEBUG_PRINTF("random sphere r=%f\n", norm3(random_inside_sphere));
	}
	
	float random_pi = (random_inside_sphere[3] - 0.5) * 2.0 * pi; // ~ U[-pi, pi]
	if (which == 0){
		DEBUG_PRINTF("random pi=%f\n", random_pi);
	}

	if (which == 0) {
		for (int i = 0; i < 3; i++)
			c->position[i] += amplitude * random_inside_sphere[i];
		return;
	}
	--which;
	if (which == 0) {
		float gr = gyration_radius(m_lig_begin, m_lig_end, atoms, m_coords_gpu, m_lig_node_origin_gpu);
		if (gr > epsilon_fl) {
			float rotation[3];
			for (int i = 0; i < 3; i++) rotation[i] = amplitude / gr * random_inside_sphere[i];
			quaternion_increment(c->orientation, rotation, epsilon_fl);
		}
		return;
	}
	--which;
	if (which < c->lig_torsion_size) { c->lig_torsion[which] = random_pi; return; }
	which -= c->lig_torsion_size;

	if (flex_torsion_size != 0) {
		if (which < flex_torsion_size) { c->flex_torsion[which] = random_pi; return; }
		which -= flex_torsion_size;
	}
}

/*  Above based on mutate_conf.cpp */

/* Below based on matrix.cpp */

// symmetric matrix_d (only half of it are stored)
typedef struct {
	float data[MAX_HESSIAN_MATRIX_D_SIZE];
	int dim;
}matrix_d;

 __device__ __forceinline__
void matrix_d_init(matrix_d* m, int dim, float fill_data) {
	m->dim = dim;
	if ((dim * (dim + 1) / 2) > MAX_HESSIAN_MATRIX_D_SIZE)DEBUG_PRINTF("\nnmatrix_d: matrix_d_init() ERROR!");
	// ((dim * (dim + 1) / 2)*sizeof(float)); // symmetric matrix_d
	for (int i = 0; i < (dim * (dim + 1) / 2); i++)m->data[i] = fill_data;
	for (int i = (dim * (dim + 1) / 2); i < MAX_HESSIAN_MATRIX_D_SIZE; i++)m->data[i] = 0;// Others will be 0
}

// as rugular 3x3 matrix_d
 __device__ __forceinline__
void mat_init(matrix_d* m, float fill_data) {
	m->dim = 3; // fixed to 3x3 matrix_d
	if (9 > MAX_HESSIAN_MATRIX_D_SIZE)DEBUG_PRINTF("\nnmatrix_d: mat_init() ERROR!");
	for (int i = 0; i < 9; i++)m->data[i] = fill_data;
}

 __device__ __forceinline__
void matrix_d_set_diagonal(matrix_d* m, float fill_data) {
	for (int i = 0; i < m->dim; i++) {
		m->data[i + i * (i + 1) / 2] = fill_data;
	}
}

// as regular matrix_d
 __device__ __forceinline__
void matrix_d_set_element(matrix_d* m, int dim, int x, int y, float fill_data) {
	m->data[x + y * dim] = fill_data;
}

 __device__ __forceinline__
void matrix_d_set_element_tri(matrix_d* m, int x, int y, float fill_data) {
	m->data[x + y*(y+1)/2] = fill_data;
}
 __device__ __forceinline__
int tri_index(int n, int i, int j) {
	if (j >= n || i > j)DEBUG_PRINTF("\nmatrix_d: tri_index ERROR!");
	return i + j * (j + 1) / 2;
}

 __device__ __forceinline__
int index_permissive(const matrix_d* m, int i, int j) {
	return (i < j) ? tri_index(m->dim, i, j) : tri_index(m->dim, j, i);
}

/* Above based on matrix_d.cpp */

/* Below based on quasi_newton.cpp */

 __device__ __forceinline__
void change_cuda_init(change_cuda_t* g, const float* ptr) {
	for (int i = 0; i < 3; i++)g->position[i] = ptr[i];
	for (int i = 0; i < 3; i++)g->orientation[i] = ptr[i + 3];
	for (int i = 0; i < MAX_NUM_OF_LIG_TORSION; i++)g->lig_torsion[i] = ptr[i + 3 + 3];
	for (int i = 0; i < MAX_NUM_OF_FLEX_TORSION; i++)g->flex_torsion[i] = ptr[i + 3 + 3 + MAX_NUM_OF_LIG_TORSION];
	g->lig_torsion_size = ptr[3 + 3 + MAX_NUM_OF_LIG_TORSION + MAX_NUM_OF_FLEX_TORSION];
}

 __device__ __forceinline__
void change_cuda_init_with_change(change_cuda_t* g_new, const change_cuda_t* g_old) {
	for (int i = 0; i < 3; i++)g_new->position[i] = g_old->position[i];
	for (int i = 0; i < 3; i++)g_new->orientation[i] = g_old->orientation[i];
	for (int i = 0; i < MAX_NUM_OF_LIG_TORSION; i++)g_new->lig_torsion[i] = g_old->lig_torsion[i];
	for (int i = 0; i < MAX_NUM_OF_FLEX_TORSION; i++)g_new->flex_torsion[i] = g_old->flex_torsion[i];
	g_new->lig_torsion_size = g_old->lig_torsion_size;
}

void print_output_type(output_type_cuda_t* x, int torsion_size) {
	for (int i = 0; i < 3; i++)DEBUG_PRINTF("\nx.position[%d] = %0.16f", i, x->position[i]);
	for (int i = 0; i < 4; i++)DEBUG_PRINTF("\nx.orientation[%d] = %0.16f", i, x->orientation[i]);
	for (int i = 0; i < torsion_size; i++)DEBUG_PRINTF("\n x.torsion[%d] = %0.16f", i, x->lig_torsion[i]);
	DEBUG_PRINTF("\n x.torsion_size = %f", x->lig_torsion_size);
	DEBUG_PRINTF("\n !!! x.e = %f\n", x->e);
}

void print_change(change_cuda_t* g, int torsion_size) {
	for (int i = 0; i < 3; i++)DEBUG_PRINTF("\ng.position[%d] = %0.16f", i, g->position[i]);
	for (int i = 0; i < 3; i++)DEBUG_PRINTF("\ng.orientation[%d] = %0.16f", i, g->orientation[i]);
	for (int i = 0; i < torsion_size; i++)DEBUG_PRINTF("\ng.torsion[%d] = %0.16f", i, g->lig_torsion[i]);
	DEBUG_PRINTF("\ng.torsion_size = %f", g->lig_torsion_size);
}

 __device__ __forceinline__
int num_atom_types(int atu) {
	switch (atu) {
	case 0: return EL_TYPE_SIZE;
	case 1: return AD_TYPE_SIZE;
	case 2: return XS_TYPE_SIZE;
	case 3: return SY_TYPE_SIZE;
	default: DEBUG_PRINTF("Kernel1:num_atom_types() ERROR!"); return INFINITY;
	}
}

 __device__ __forceinline__
void elementwise_product(float* out, const float* a, const float* b) {
	out[0] = a[0] * b[0];
	out[1] = a[1] * b[1];
	out[2] = a[2] * b[2];
}

 __device__ __forceinline__
float elementwise_product_sum(const float* a, const float* b) {
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

 __device__ __forceinline__
float access_m_data(float* m_data, int m_i, int m_j, int i, int j, int k) {
	return m_data[i + m_i * (j + m_j * k)];
}

__device__ __forceinline__
bool not_max_gpu(float x) {
	return (x < 0.1 * INFINITY); /* Problem: replace max_fl with INFINITY? */
}

 __device__ __forceinline__
void curl_with_deriv(float* e, float* deriv, float v, const float epsilon_fl) {
	if (*e > 0 && not_max_gpu(v)) {
		float tmp = (v < epsilon_fl) ? 0 : (v / (v + *e));
		*e *= tmp;
		for (int i = 0; i < 3; i++)deriv[i] *= pow(tmp, 2);
	}
}

 __device__ __forceinline__
void curl_without_deriv(float* e, float v, const float epsilon_fl) {
	if (*e > 0 && not_max_gpu(v)) {
		float tmp = (v < epsilon_fl) ? 0 : (v / (v + *e));
		*e *= tmp;
	}
}

 __device__ __forceinline__
float g_evaluate(	grid_cuda_t*	g,
					const				float*		m_coords,			/* double[3] */
					const				float		slope,				/* double */
					const				float		v,					/* double */
										float*		deriv,				/* double[3] */
					const				float		epsilon_fl
) {
	int m_i = g->m_i;
	int m_j = g->m_j;
	int m_k = g->m_k;
	if(m_i * m_j * m_k == 0)DEBUG_PRINTF("\nkernel2: g_evaluate ERROR!#1");
	float tmp_vec[3] = { m_coords[0] - g->m_init[0],m_coords[1] - g->m_init[1] ,m_coords[2] - g->m_init[2] };
	float tmp_vec2[3] = { g->m_factor[0],g->m_factor[1] ,g->m_factor[2] };
	float s[3];
	elementwise_product(s, tmp_vec, tmp_vec2);

	float miss[3] = { 0,0,0 };
	int region[3];
	int a[3];
	int m_data_dims[3] = { m_i,m_j,m_k };
	for (int i = 0; i < 3; i++){
		if (s[i] < 0) {
			miss[i] = -s[i];
			region[i] = -1;
			a[i] = 0;
			s[i] = 0;
		}
		else if (s[i] >= g->m_dim_fl_minus_1[i]) {
			miss[i] = s[i] - g->m_dim_fl_minus_1[i];
			region[i] = 1;
			if (m_data_dims[i] < 2)DEBUG_PRINTF("\nKernel2: g_evaluate ERROR!#2");
			a[i] = m_data_dims[i] - 2;
			s[i] = 1;
		}
		else {
			region[i] = 0;
			a[i] = (int)s[i];
			s[i] -= a[i];
		}
		if (s[i] < 0)
            DEBUG_PRINTF("\nKernel2: g_evaluate ERROR!#3");
		if (s[i] > 1)
            DEBUG_PRINTF("\nKernel2: g_evaluate ERROR!#4");
		if (a[i] < 0)
            DEBUG_PRINTF("\nKernel2: g_evaluate ERROR!#5");
		if (a[i] + 1 >= m_data_dims[i])DEBUG_PRINTF("\nKernel2: g_evaluate ERROR!#5");
	}

	float tmp_m_factor_inv[3] = { g->m_factor_inv[0],g->m_factor_inv[1],g->m_factor_inv[2] };
	const float penalty = slope * elementwise_product_sum(miss, tmp_m_factor_inv);
	if (penalty <= -epsilon_fl)DEBUG_PRINTF("\nKernel2: g_evaluate ERROR!#6");

	const int x0 = a[0];
	const int y0 = a[1];
	const int z0 = a[2];

	const int x1 = x0 + 1;
	const int y1 = y0 + 1;
	const int z1 = z0 + 1;

	const float f000 = access_m_data(g->m_data, m_i, m_j, x0, y0, z0);
	const float f100 = access_m_data(g->m_data, m_i, m_j, x1, y0, z0);
	const float f010 = access_m_data(g->m_data, m_i, m_j, x0, y1, z0);
	const float f110 = access_m_data(g->m_data, m_i, m_j, x1, y1, z0);
	const float f001 = access_m_data(g->m_data, m_i, m_j, x0, y0, z1);
	const float f101 = access_m_data(g->m_data, m_i, m_j, x1, y0, z1);
	const float f011 = access_m_data(g->m_data, m_i, m_j, x0, y1, z1);
	const float f111 = access_m_data(g->m_data, m_i, m_j, x1, y1, z1);


	const float x = s[0];
	const float y = s[1];
	const float z = s[2];

	const float mx = 1 - x;
	const float my = 1 - y;
	const float mz = 1 - z;

	float f =
		f000 * mx * my * mz +
		f100 * x  * my * mz +
		f010 * mx * y  * mz +
		f110 * x  * y  * mz +
		f001 * mx * my * z	+
		f101 * x  * my * z	+
		f011 * mx * y  * z	+
		f111 * x  * y  * z  ;

	if (deriv) {
		const float x_g =
			f000 * (-1) * my * mz +
			f100 *   1  * my * mz +
			f010 * (-1) * y  * mz +
			f110 *	 1  * y  * mz +
			f001 * (-1) * my * z  +
			f101 *   1  * my * z  +
			f011 * (-1) * y  * z  +
			f111 *   1  * y  * z  ;


		const float y_g =
			f000 * mx * (-1) * mz +
			f100 * x  * (-1) * mz +
			f010 * mx *   1  * mz +
			f110 * x  *   1  * mz +
			f001 * mx * (-1) * z  +
			f101 * x  * (-1) * z  +
			f011 * mx *   1  * z  +
			f111 * x  *   1  * z  ;


		const float z_g =
			f000 * mx * my * (-1) +
			f100 * x  * my * (-1) +
			f010 * mx * y  * (-1) +
			f110 * x  * y  * (-1) +
			f001 * mx * my *   1  +
			f101 * x  * my *   1  +
			f011 * mx * y  *   1  +
			f111 * x  * y  *   1  ;

		float gradient[3] = { x_g, y_g, z_g };

		curl_with_deriv(&f, gradient, v, epsilon_fl);

		float gradient_everywhere[3];

		for (int i = 0; i < 3; i++) {
			gradient_everywhere[i] = ((region[i] == 0) ? gradient[i] : 0);
			deriv[i] = g->m_factor[i] * gradient_everywhere[i] + slope * region[i];
		}
		return f + penalty;
	}
	else {  /* none valid pointer */
		DEBUG_PRINTF("\nKernel2: g_evaluate ERROR!#7");
		curl_without_deriv(&f, v, epsilon_fl);
		return f + penalty;
	}
}

 __device__ __forceinline__
float ig_eval_deriv(						output_type_cuda_t*		x,
											change_cuda_t*			g,
						const				float				v,
									ig_cuda_t*				ig_cuda_gpu,
											m_cuda_t*				m_cuda_gpu,
						const				float				epsilon_fl
) {
	float e = 0;
	int nat = num_atom_types(ig_cuda_gpu->atu);
	for (int i = 0; i < m_cuda_gpu->m_num_movable_atoms; i++) {
		int t = m_cuda_gpu->atoms[i].types[ig_cuda_gpu->atu];
		if (t >= nat) {
			for (int j = 0; j < 3; j++)m_cuda_gpu->minus_forces.coords[i][j] = 0;
			continue;
		}
		float deriv[3];

		e = e + g_evaluate(&ig_cuda_gpu->grids[t], m_cuda_gpu->m_coords.coords[i], ig_cuda_gpu->slope, v, deriv, epsilon_fl);

		for (int j = 0; j < 3; j++) m_cuda_gpu->minus_forces.coords[i][j] = deriv[j];
	}
	return e;
}

 __device__ __forceinline__
void quaternion_to_r3(const float* q, float* orientation_m) {
	/* Omit assert(quaternion_is_normalized(q)); */
	const float a = q[0];
	const float b = q[1];
	const float c = q[2];
	const float d = q[3];

	const float aa = a * a;
	const float ab = a * b;
	const float ac = a * c;
	const float ad = a * d;
	const float bb = b * b;
	const float bc = b * c;
	const float bd = b * d;
	const float cc = c * c;
	const float cd = c * d;
	const float dd = d * d;

	/* Omit assert(eq(aa + bb + cc + dd, 1)); */
	matrix_d tmp;
	mat_init(&tmp, 0); /* matrix_d with fixed dimension 3(here we treate this as a regular matrix_d(not triangular matrix_d!)) */

	matrix_d_set_element(&tmp, 3, 0, 0,		(aa + bb - cc - dd)	);
	matrix_d_set_element(&tmp, 3, 0, 1, 2 *	(-ad + bc)			);
	matrix_d_set_element(&tmp, 3, 0, 2, 2 *	(ac + bd)			);

	matrix_d_set_element(&tmp, 3, 1, 0, 2 *	(ad + bc)			);
	matrix_d_set_element(&tmp, 3, 1, 1,		(aa - bb + cc - dd)	);
	matrix_d_set_element(&tmp, 3, 1, 2, 2 *	(-ab + cd)			);

	matrix_d_set_element(&tmp, 3, 2, 0, 2 *	(-ac + bd)			);
	matrix_d_set_element(&tmp, 3, 2, 1, 2 *	(ab + cd)			);
	matrix_d_set_element(&tmp, 3, 2, 2,		(aa - bb - cc + dd)	);

	for (int i = 0; i < 9; i++) orientation_m[i] = tmp.data[i];
}

 __device__ __forceinline__
void local_to_lab_direction(			float* out,
									const	float* local_direction,
									const	float* orientation_m
) {
	out[0] =	orientation_m[0] * local_direction[0] +
				orientation_m[3] * local_direction[1] +
				orientation_m[6] * local_direction[2];
	out[1] =	orientation_m[1] * local_direction[0] +
				orientation_m[4] * local_direction[1] +
				orientation_m[7] * local_direction[2];
	out[2] =	orientation_m[2] * local_direction[0] +
				orientation_m[5] * local_direction[1] +
				orientation_m[8] * local_direction[2];
}

 __device__ __forceinline__
void local_to_lab(						float*		out,
							const				float*		origin,
							const				float*		local_coords,
							const				float*		orientation_m
) {
	out[0] = origin[0] + (	orientation_m[0] * local_coords[0] +
							orientation_m[3] * local_coords[1] +
							orientation_m[6] * local_coords[2]
							);
	out[1] = origin[1] + (	orientation_m[1] * local_coords[0] +
							orientation_m[4] * local_coords[1] +
							orientation_m[7] * local_coords[2]
							);
	out[2] = origin[2] + (	orientation_m[2] * local_coords[0] +
							orientation_m[5] * local_coords[1] +
							orientation_m[8] * local_coords[2]
							);
}

 __device__ __forceinline__
void angle_to_quaternion2(				float*		out,
									const		float*		axis,
												float		angle
) {
	normalize_angle(&angle);
	float c = cos(angle / 2);
	float s = sin(angle / 2);
	out[0] = c;
	out[1] = s * axis[0];
	out[2] = s * axis[1];
	out[3] = s * axis[2];
}

__device__ __forceinline__
void set(	const				output_type_cuda_t* x,
								rigid_cuda_t*		lig_rigid_gpu,
								m_coords_cuda_t*		m_coords_gpu,
			const				atom_cuda_t*		atoms,
			const				int				m_num_movable_atoms,
			const				float			epsilon_fl
) {

	for (int i = 0; i < 3; i++) lig_rigid_gpu->origin[0][i] = x->position[i];
	for (int i = 0; i < 4; i++) lig_rigid_gpu->orientation_q[0][i] = x->orientation[i];
	quaternion_to_r3(lig_rigid_gpu->orientation_q[0], lig_rigid_gpu->orientation_m[0]); /* set orientation_m */

	int begin = lig_rigid_gpu->atom_range[0][0];
	int end =	lig_rigid_gpu->atom_range[0][1];
	for (int i = begin; i < end; i++) {
		local_to_lab(m_coords_gpu->coords[i], lig_rigid_gpu->origin[0], atoms[i].coords, lig_rigid_gpu->orientation_m[0]);
	}
	/* ************* end node.set_conf ************* */

	/* ************* branches_set_conf ************* */
	/* update nodes in depth-first order */
	for (int current = 1; current < lig_rigid_gpu->num_children + 1; current++) { /* current starts from 1 (namely starts from first child node) */
		int parent = lig_rigid_gpu->parent[current];
		float torsion = x->lig_torsion[current - 1]; /* torsions are all related to child nodes */
		local_to_lab(	lig_rigid_gpu->origin[current],
						lig_rigid_gpu->origin[parent],
						lig_rigid_gpu->relative_origin[current],
						lig_rigid_gpu->orientation_m[parent]
						);
		local_to_lab_direction(	lig_rigid_gpu->axis[current],
								lig_rigid_gpu->relative_axis[current],
								lig_rigid_gpu->orientation_m[parent]
								);
		float tmp[4];
		float parent_q[4] = {	lig_rigid_gpu->orientation_q[parent][0],
								lig_rigid_gpu->orientation_q[parent][1] ,
								lig_rigid_gpu->orientation_q[parent][2] ,
								lig_rigid_gpu->orientation_q[parent][3] };
		float current_axis[3] = {	lig_rigid_gpu->axis[current][0],
									lig_rigid_gpu->axis[current][1],
									lig_rigid_gpu->axis[current][2] };

		angle_to_quaternion2(tmp, current_axis, torsion);
		angle_to_quaternion_multi(tmp, parent_q);
		quaternion_normalize_approx(tmp, epsilon_fl);

		for (int i = 0; i < 4; i++) lig_rigid_gpu->orientation_q[current][i] = tmp[i]; /* set orientation_q */
		quaternion_to_r3(lig_rigid_gpu->orientation_q[current], lig_rigid_gpu->orientation_m[current]); /* set orientation_m */

		/* set coords */
		begin = lig_rigid_gpu->atom_range[current][0];
		end =	lig_rigid_gpu->atom_range[current][1];
		for (int i = begin; i < end; i++) {
			local_to_lab(m_coords_gpu->coords[i], lig_rigid_gpu->origin[current], atoms[i].coords, lig_rigid_gpu->orientation_m[current]);
		}
	}
	/* ************* end branches_set_conf ************* */
}

 __device__ __forceinline__
void p_eval_deriv(						float*		out,
										int			type_pair_index,
										float		r2,
									p_cuda_t*		p_cuda_gpu,
					const				float		epsilon_fl
) {
	const float cutoff_sqr = p_cuda_gpu->m_cutoff_sqr;
	if(r2 > cutoff_sqr) DEBUG_PRINTF("\nkernel2: p_eval_deriv() ERROR!, r2 > Cutoff_sqr, r2=%f, cutoff_sqr=%f", r2, cutoff_sqr);

	p_m_data_cuda_t* tmp = &p_cuda_gpu->m_data[type_pair_index];
	float r2_factored = tmp->factor * r2;
	int i1 = (int)(r2_factored);
	int i2 = i1 + 1;
	float rem = r2_factored - i1;
	if (rem < -epsilon_fl)DEBUG_PRINTF("\nkernel2: p_eval_deriv() ERROR!");
	if (rem >= 1 + epsilon_fl)DEBUG_PRINTF("\nkernel2: p_eval_deriv() ERROR!");
	float p1[2] = { tmp->smooth[i1][0], tmp->smooth[i1][1] };
	if (i1 >= SMOOTH_SIZE) p1[0] = p1[1] = 0;
	float p2[2] = { tmp->smooth[i2][0], tmp->smooth[i2][1] };
	if (i2 >= SMOOTH_SIZE) p2[0] = p2[1] = 0;
	float e = p1[0] + rem * (p2[0] - p1[0]);
	float dor = p1[1] + rem * (p2[1] - p1[1]);
	out[0] = e;
	out[1] = dor;
}

 __device__ __forceinline__
void curl(float* e, float* deriv, float v, const float epsilon_fl) {
	if (*e > 0 && not_max_gpu(v)) {
		float tmp = (v < epsilon_fl) ? 0 : (v / (v + *e));
		(*e) = tmp * (*e);
		for (int i = 0; i < 3; i++)deriv[i] = deriv[i] * (tmp * tmp);
	}
}

 __device__ __forceinline__
float eval_interacting_pairs_deriv(	p_cuda_t*			p_cuda_gpu,
									const				float			v,
									const				lig_pairs_cuda_t*   pairs,
									const			 	m_coords_cuda_t*		m_coords,
									m_minus_forces_t* 	minus_forces,
									const				float			epsilon_fl
) {
	float e = 0;
	for (int i = 0; i < pairs->num_pairs; i++) {
		const int ip[3] = { pairs->type_pair_index[i], pairs->a[i] ,pairs->b[i] };
		int index = pairs->a[i] + pairs->b[i] * (pairs->b[i] + 1) / 2;
		float coords_b[3] = { m_coords->coords[ip[2]][0], m_coords->coords[ip[2]][1], m_coords->coords[ip[2]][2] };
		float coords_a[3] = { m_coords->coords[ip[1]][0], m_coords->coords[ip[1]][1], m_coords->coords[ip[1]][2] };
		float r[3] = { coords_b[0] - coords_a[0], coords_b[1] - coords_a[1] ,coords_b[2] - coords_a[2] };
		float r2 = r[0] * r[0] + r[1] * r[1] + r[2] * r[2];

		if (r2 < p_cuda_gpu->m_cutoff_sqr) {
			float tmp[2];
			p_eval_deriv(tmp, index, r2, p_cuda_gpu, epsilon_fl);
			float force[3] = { r[0] * tmp[1], r[1] * tmp[1] ,r[2] * tmp[1] };
			curl(&tmp[0], force, v, epsilon_fl);
			e += tmp[0];
			for (int j = 0; j < 3; j++)minus_forces->coords[ip[1]][j] -= force[j];
			for (int j = 0; j < 3; j++)minus_forces->coords[ip[2]][j] += force[j];
		}
	}
	return e;
}

 __device__ __forceinline__
void product(float* res, const float*a,const float*b) {
	res[0] = a[1] * b[2] - a[2] * b[1];
	res[1] = a[2] * b[0] - a[0] * b[2];
	res[2] = a[0] * b[1] - a[1] * b[0];
}

 __device__ __forceinline__
void POT_deriv(	const					m_minus_forces_t* minus_forces,
				const					rigid_cuda_t*		lig_rigid_gpu,
				const					m_coords_cuda_t*		m_coords,
										change_cuda_t*		g
) {
	int num_torsion = lig_rigid_gpu->num_children;
	int num_rigid = num_torsion + 1;
	float position_derivative_tmp[MAX_NUM_OF_RIGID][3];
	float position_derivative[MAX_NUM_OF_RIGID][3];
	float orientation_derivative_tmp[MAX_NUM_OF_RIGID][3];
	float orientation_derivative[MAX_NUM_OF_RIGID][3];
	float torsion_derivative[MAX_NUM_OF_RIGID]; /* torsion_derivative[0] has no meaning(root node has no torsion) */

	for (int i = 0; i < num_rigid; i++) {
		int begin = lig_rigid_gpu->atom_range[i][0];
		int end = lig_rigid_gpu->atom_range[i][1];
		for (int k = 0; k < 3; k++)position_derivative_tmp[i][k] = 0;
		for (int k = 0; k < 3; k++)orientation_derivative_tmp[i][k] = 0;
		for (int j = begin; j < end; j++) {
			for (int k = 0; k < 3; k++)position_derivative_tmp[i][k] += minus_forces->coords[j][k];

			float tmp1[3] = {	m_coords->coords[j][0] - lig_rigid_gpu->origin[i][0],
								m_coords->coords[j][1] - lig_rigid_gpu->origin[i][1],
								m_coords->coords[j][2] - lig_rigid_gpu->origin[i][2] };
			float tmp2[3] = {  minus_forces->coords[j][0],
								minus_forces->coords[j][1],
								minus_forces->coords[j][2] };
			float tmp3[3];
			product(tmp3, tmp1, tmp2);
			for (int k = 0; k < 3; k++)
                orientation_derivative_tmp[i][k] += tmp3[k];
		}
	}

	/* position_derivative  */
	for (int i = num_rigid - 1; i >= 0; i--) { /* from bottom to top */
		for (int k = 0; k < 3; k++)position_derivative[i][k] = position_derivative_tmp[i][k];
		/* looking for chidren node */
		for (int j = 0; j < num_rigid; j++) {
			if (lig_rigid_gpu->children_map[i][j] == true) {
				for (int k = 0; k < 3; k++)position_derivative[i][k] += position_derivative[j][k]; /* self+children node */
			}
		}
	}

	/* orientation_derivetive */
	for (int i = num_rigid - 1; i >= 0; i--) { /* from bottom to top */
		for (int k = 0; k < 3; k++)orientation_derivative[i][k] = orientation_derivative_tmp[i][k];
		/* looking for chidren node */
		for (int j = 0; j < num_rigid; j++) {
			if (lig_rigid_gpu->children_map[i][j] == true) { /* self + children node + product */
				for (int k = 0; k < 3; k++)orientation_derivative[i][k] += orientation_derivative[j][k];
				float product_out[3];
				float origin_temp[3] = {	lig_rigid_gpu->origin[j][0] - lig_rigid_gpu->origin[i][0],
											lig_rigid_gpu->origin[j][1] - lig_rigid_gpu->origin[i][1],
											lig_rigid_gpu->origin[j][2] - lig_rigid_gpu->origin[i][2] };
				product(product_out, origin_temp, position_derivative[j]);
				for (int k = 0; k < 3; k++)orientation_derivative[i][k] += product_out[k];
			}
		}
	}

	/* torsion_derivative */
	for (int i = num_rigid - 1; i >= 0; i--) {
		float sum = 0;
		for (int j = 0; j < 3; j++) sum += orientation_derivative[i][j] * lig_rigid_gpu->axis[i][j];
		torsion_derivative[i] = sum;
	}

	for (int k = 0; k < 3; k++)	g->position[k] = position_derivative[0][k];
	for (int k = 0; k < 3; k++) g->orientation[k] = orientation_derivative[0][k];
	for (int k = 0; k < num_torsion; k++) g->lig_torsion[k] = torsion_derivative[k + 1];
}

 __device__ __forceinline__
float m_eval_deriv(					output_type_cuda_t*		c,
										change_cuda_t*			g,
										m_cuda_t*				m_cuda_gpu,
								p_cuda_t*				p_cuda_gpu,
								ig_cuda_t*				ig_cuda_gpu,
					const	float*				v,
					const				float				epsilon_fl
) {
	// check set args
	set(c, &m_cuda_gpu->ligand.rigid, &m_cuda_gpu->m_coords, m_cuda_gpu->atoms, m_cuda_gpu->m_num_movable_atoms, epsilon_fl);

	float e = 0;
	e = ig_eval_deriv(	c,
								g,
								v[1],
								ig_cuda_gpu,
								m_cuda_gpu,
								epsilon_fl
							);
	e += eval_interacting_pairs_deriv(	p_cuda_gpu,
										v[0],
										&m_cuda_gpu->ligand.pairs,
										&m_cuda_gpu->m_coords,
										&m_cuda_gpu->minus_forces,
										epsilon_fl
									);
	// should add derivs for glue, other and inter pairs
	POT_deriv(&m_cuda_gpu->minus_forces, &m_cuda_gpu->ligand.rigid, &m_cuda_gpu->m_coords, g);

	return e;
}


 __device__ __forceinline__
float find_change_index_read(const change_cuda_t* g, int index) {
	if (index < 3) return g->position[index];
	index -= 3;
	if (index < 3) return g->orientation[index];
	index -= 3;
	if (index < g->lig_torsion_size) return g->lig_torsion[index];
	DEBUG_PRINTF("\nKernel2:find_change_index_read() ERROR!"); /* Shouldn't be here */
}

 __device__ __forceinline__
void find_change_index_write(change_cuda_t* g, int index, float data) {
	if (index < 3) { g->position[index] = data; return; }
	index -= 3;
	if (index < 3) { g->orientation[index] = data; return; }
	index -= 3;
	if (index < g->lig_torsion_size) { g->lig_torsion[index] = data; return; }
	DEBUG_PRINTF("\nKernel2:find_change_index_write() ERROR!"); /* Shouldn't be here */
}

 __device__ __forceinline__
void minus_mat_vec_product(	const		matrix_d*		h,
							const		change_cuda_t*	in,
										change_cuda_t*  out
) {
	int n = h->dim;
	for (int i = 0; i < n; i++) {
		float sum = 0;
		for (int j = 0; j < n; j++) {
			sum += h->data[index_permissive(h, i, j)] * find_change_index_read(in, j);
		}
		find_change_index_write(out, i, -sum);
	}
}


 __device__ __forceinline__
float scalar_product(	const	change_cuda_t*			a,
								const	change_cuda_t*			b,
								int							n
) {
	float tmp = 0;
	for (int i = 0; i < n; i++) {
		tmp += find_change_index_read(a, i) * find_change_index_read(b, i);
	}
	return tmp;
}

 __device__ __forceinline__
float line_search(					 	m_cuda_t*				m_cuda_gpu,
								p_cuda_t*				p_cuda_gpu,
								ig_cuda_t*				ig_cuda_gpu,
										int					n,
					const				output_type_cuda_t*		x,
					const				change_cuda_t*			g,
					const				float				f0,
					const				change_cuda_t*			p,
										output_type_cuda_t*		x_new,
										change_cuda_t*			g_new,
										float*				f1,
					const				float				epsilon_fl,
					const	float*				hunt_cap
) {
	const float c0 = 0.0001;
	const int max_trials = 10;
	const float multiplier = 0.5;
	float alpha = 1;

	const float pg = scalar_product(p, g, n);

	for (int trial = 0; trial < max_trials; trial++) {

		output_type_cuda_init_with_output(x_new, x);
		output_type_cuda_increment(x_new, p, alpha, epsilon_fl);
		*f1 =  m_eval_deriv(x_new,
							g_new,
							m_cuda_gpu,
							p_cuda_gpu,
							ig_cuda_gpu,
							hunt_cap,
							epsilon_fl
							);
		if (*f1 - f0 < c0 * alpha * pg)
			break;
		alpha *= multiplier;
	}
	return alpha;
}

 __device__ __forceinline__
bool bfgs_update(			matrix_d*			h,
					const	change_cuda_t*		p,
					const	change_cuda_t*		y,
					const	float			alpha,
					const	float			epsilon_fl
) {

	const float yp = scalar_product(y, p, h->dim);

	if (alpha * yp < epsilon_fl) return false;
	change_cuda_t minus_hy;
	change_cuda_init_with_change(&minus_hy, y);
	minus_mat_vec_product(h, y, &minus_hy);
	const float yhy = -scalar_product(y, &minus_hy, h->dim);
	const float r = 1 / (alpha * yp);
	const int n = 6 + p->lig_torsion_size;

	for (int i = 0; i < n; i++) {
		for (int j = i; j < n; j++) {
			float tmp = alpha * r * (find_change_index_read(&minus_hy, i) * find_change_index_read(p, j)
									+ find_change_index_read(&minus_hy, j) * find_change_index_read(p, i)) +
									+alpha * alpha * (r * r * yhy + r) * find_change_index_read(p, i) * find_change_index_read(p, j);

			h->data[i + j * (j + 1) / 2] += tmp;
		}
	}

	return true;
}


 __device__ __forceinline__
void bfgs(					output_type_cuda_t*			x,
								change_cuda_t*			g,
								m_cuda_t*				m_cuda_gpu,
						p_cuda_t*				p_cuda_gpu,
						ig_cuda_t*				ig_cuda_gpu,
			const	float*				hunt_cap,
			const				float				epsilon_fl,
			const				int					max_steps
)
{
	int n = 3 + 3 + x->lig_torsion_size; /* the dimensions of matirx */

	matrix_d h;
	matrix_d_init(&h, n, 0);
	matrix_d_set_diagonal(&h, 1);

	change_cuda_t g_new;
	change_cuda_init_with_change(&g_new, g);

	output_type_cuda_t x_new;
	output_type_cuda_init_with_output(&x_new, x);

	float f0 = m_eval_deriv(	x,
								g,
								m_cuda_gpu,
								p_cuda_gpu,
								ig_cuda_gpu,
								hunt_cap,
								epsilon_fl
							);

	float f_orig = f0;
	/* Init g_orig, x_orig */
	change_cuda_t g_orig;
	change_cuda_init_with_change(&g_orig, g);
	output_type_cuda_t x_orig;
	output_type_cuda_init_with_output(&x_orig, x);
	/* Init p */
	change_cuda_t p;
	change_cuda_init_with_change(&p, g);

	for (int step = 0; step < max_steps; step++) {

		minus_mat_vec_product(&h, g, &p);
		float f1 = 0;

		const float alpha = line_search(	m_cuda_gpu,
											p_cuda_gpu,
											ig_cuda_gpu,
											n,
											x,
											g,
											f0,
											&p,
											&x_new,
											&g_new,
											&f1,
											epsilon_fl,
											hunt_cap
										);

		change_cuda_t y;
		change_cuda_init_with_change(&y, &g_new);
		/* subtract_change */
		for (int i = 0; i < n; i++) {
			float tmp = find_change_index_read(&y, i) - find_change_index_read(g, i);
			find_change_index_write(&y, i, tmp);
		}
		// f_values[step + 1] = f1;
		f0 = f1;
		output_type_cuda_init_with_output(x, &x_new);
		if (!(sqrt(scalar_product(g, g, n)) >= 1e-5))break;
		change_cuda_init_with_change(g, &g_new);

		if (step == 0) {
			float yy = scalar_product(&y, &y, n);
			if (fabs(yy) > epsilon_fl) {
				matrix_d_set_diagonal(&h, alpha * scalar_product(&y, &p, n) / yy);
			}
		}

		bool h_updated = bfgs_update(&h, &p, &y, alpha, epsilon_fl);
	}

	if (!(f0 <= f_orig)) {
		f0 = f_orig;
		output_type_cuda_init_with_output(x, &x_orig);
		change_cuda_init_with_change(g, &g_orig);
	}

	// write output_type_cuda energy
	x->e = f0;
}


/* Above based on quasi_newton.cpp */

/* Below is monte-carlo kernel, based on kernel.cl*/

 __device__ __forceinline__
void m_cuda_init_with_m_cuda(const m_cuda_t* m_cuda_old, m_cuda_t* m_cuda_new) {
	memcpy(m_cuda_new, m_cuda_old, MAX_NUM_OF_ATOMS *  sizeof(atom_cuda_t));
	m_cuda_new->m_coords = m_cuda_old->m_coords;
	m_cuda_new->minus_forces = m_cuda_old->minus_forces;
	m_cuda_new->ligand = m_cuda_old->ligand;
	m_cuda_new->m_num_movable_atoms = m_cuda_old->m_num_movable_atoms;
}


 __device__ __forceinline__
void get_heavy_atom_movable_coords(output_type_cuda_t* tmp, const m_cuda_t* m_cuda_gpu) {
	int counter = 0;
	for (int i = 0; i < m_cuda_gpu->m_num_movable_atoms; i++) {
		if (m_cuda_gpu->atoms[i].types[0] != EL_TYPE_H) {
			for (int j = 0; j < 3; j++)tmp->coords[counter][j] = m_cuda_gpu->m_coords.coords[i][j];
			counter++;
		}
		else {
			// DEBUG_PRINTF("\n P2: removed H atom coords in get_heavy_atom_movable_coords()!");
		}
	}
	/* assign 0 for others */
	for (int i = counter; i < MAX_NUM_OF_ATOMS; i++) {
		for (int j = 0; j < 3; j++)tmp->coords[i][j] = 0;
	}
}

 __device__ __forceinline__
float generate_n(const float* pi_map, const int step) {
	return fabs(pi_map[step]) / M_PI;
}

 __device__ __forceinline__
bool metropolis_accept(float old_f, float new_f, float temperature, float n) {
	if (new_f < old_f)return true;
	const float acceptance_probability = exp((old_f - new_f) / temperature);
	return n < acceptance_probability;
}

 __device__ __forceinline__
void write_back(output_type_cuda_t* results, const output_type_cuda_t* best_out) {
	for (int i = 0; i < 3; i++)results->position[i] = best_out->position[i];
	for (int i = 0; i < 4; i++)results->orientation[i] = best_out->orientation[i];
	for (int i = 0; i < MAX_NUM_OF_LIG_TORSION; i++)results->lig_torsion[i] = best_out->lig_torsion[i];
	for (int i = 0; i < MAX_NUM_OF_FLEX_TORSION; i++)results->flex_torsion[i] = best_out->flex_torsion[i];
	results->lig_torsion_size = best_out->lig_torsion_size;
	results->e = best_out->e;
	for (int i = 0; i < MAX_NUM_OF_ATOMS; i++) {
		for (int j = 0; j < 3; j++) {
			results->coords[i][j] = best_out->coords[i][j];
		}
	}
}
// MAX_THREADS_PER_BLOCK and MIN_BLOCKS_PER_MP should be adjusted according to the profiling results 
#define MAX_THREADS_PER_BLOCK 32
#define MIN_BLOCKS_PER_MP     32
__global__
__launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)
void kernel(	m_cuda_t*			m_cuda_global,
				ig_cuda_t*			ig_cuda_gpu,
				p_cuda_t*			p_cuda_gpu,
				float*				rand_molec_struc_gpu,
				float*				best_e_gpu,
				int					bfgs_max_steps,
				float				mutation_amplitude,
				hiprandStatePhilox4_32_10_t* states, 
				unsigned long long seed,
				float				epsilon_fl,
				float*				hunt_cap_gpu,
				float*				authentic_v_gpu,
				output_type_cuda_t*	results,
				int					search_depth,
				int					num_of_ligands,
				int 				threads_per_ligand,
				bool				multi_bias
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float best_e = INFINITY;

	if (idx < num_of_ligands * threads_per_ligand)
	{
		//if (idx % 100 == 0)DEBUG_PRINTF("\nThread %d START", idx);
		output_type_cuda_t tmp; // private memory, shared only in work item
		change_cuda_t g;
		m_cuda_t m_cuda_gpu;
		// update pointer to get correct ligand data
		output_type_cuda_init(&tmp, rand_molec_struc_gpu + idx * (SIZE_OF_MOLEC_STRUC / sizeof(float)));
		hiprand_init(seed, idx, 0, &states[idx]);
		m_cuda_init_with_m_cuda(m_cuda_global + idx / threads_per_ligand, &m_cuda_gpu);
		if (multi_bias){
			ig_cuda_gpu = ig_cuda_gpu + idx / threads_per_ligand;
		}
		if (m_cuda_gpu.m_num_movable_atoms == -1){
			return;
		}
		p_cuda_gpu = p_cuda_gpu + idx / threads_per_ligand;

		g.lig_torsion_size = tmp.lig_torsion_size;
		// BFGS
		output_type_cuda_t best_out;
		output_type_cuda_t candidate;

		for (int step = 0; step < search_depth; step++) {
			output_type_cuda_init_with_output(&candidate, &tmp);
			mutate_conf_cuda(bfgs_max_steps, &candidate, &states[idx],
				m_cuda_gpu.ligand.begin, m_cuda_gpu.ligand.end, m_cuda_gpu.atoms,
				&m_cuda_gpu.m_coords, m_cuda_gpu.ligand.rigid.origin[0], epsilon_fl, mutation_amplitude);
			bfgs(&candidate, &g, &m_cuda_gpu, p_cuda_gpu, ig_cuda_gpu, hunt_cap_gpu, epsilon_fl, bfgs_max_steps);
			// n ~ U[0,1]
			float n = hiprand_uniform(&states[idx]);

			// if (idx == 0)
			// 	DEBUG_PRINTF("metropolis_accept tmp.e=%f, candidate.e=%f, n=%f\n", tmp.e, candidate.e, n);

			if (step == 0 || metropolis_accept(tmp.e, candidate.e, 1.2, n)) {
				output_type_cuda_init_with_output(&tmp, &candidate);
				set(&tmp, &m_cuda_gpu.ligand.rigid, &m_cuda_gpu.m_coords,
					m_cuda_gpu.atoms, m_cuda_gpu.m_num_movable_atoms, epsilon_fl);
				if (tmp.e < best_e) {
					bfgs(	&tmp,
							&g,
							&m_cuda_gpu,
							p_cuda_gpu,
							ig_cuda_gpu,
							authentic_v_gpu,
							epsilon_fl,
							bfgs_max_steps
					);
					// set
					if (tmp.e < best_e) {
						set(&tmp, &m_cuda_gpu.ligand.rigid, &m_cuda_gpu.m_coords,
							m_cuda_gpu.atoms, m_cuda_gpu.m_num_movable_atoms, epsilon_fl);
						output_type_cuda_init_with_output(&best_out, &tmp);
						get_heavy_atom_movable_coords(&best_out, &m_cuda_gpu); // get coords
						best_e = tmp.e;
					}

				}
			}

		}
		// write the best conformation back to CPU // FIX?? should add more
		write_back(results + idx, &best_out);
		// if (idx % 100 == 0) DEBUG_PRINTF("\nThread %d FINISH", idx);
	}
}

/* Above based on kernel.cl */

/* Below based on monte-carlo.cpp */

// #ifdef ENABLE_CUDA

std::vector<output_type> monte_carlo::cuda_to_vina(output_type_cuda_t results_ptr[], int thread) const {
	// DEBUG_PRINTF("entering cuda_to_vina\n");
	std::vector<output_type> results_vina;
	for (int i = 0; i < thread; ++i){
		output_type_cuda_t results = results_ptr[i];
		conf tmp_c;
		tmp_c.ligands.resize(1);
		// Position
		for (int j = 0; j < 3; j++)tmp_c.ligands[0].rigid.position[j] = results.position[j];
		// Orientation
		qt q(results.orientation[0], results.orientation[1], results.orientation[2], results.orientation[3]);
		tmp_c.ligands[0].rigid.orientation = q;
		output_type tmp_vina(tmp_c, results.e);
		// torsion
		for (int j = 0; j < results.lig_torsion_size; j++) tmp_vina.c.ligands[0].torsions.push_back(results.lig_torsion[j]);
		// coords
		for (int j = 0; j < MAX_NUM_OF_ATOMS; j++) {
			vec v_tmp(results.coords[j][0], results.coords[j][1], results.coords[j][2]);
			if (v_tmp[0] * v_tmp[1] * v_tmp[2] != 0) tmp_vina.coords.push_back(v_tmp);
		}
		results_vina.push_back(tmp_vina);
	}
	return results_vina;
}

__host__
void monte_carlo::operator()(std::vector<model>& m_gpu, std::vector<output_container>& out_gpu, std::vector<precalculate_byatom> & p_gpu,
				triangular_matrix_cuda_t *m_data_list_gpu, const igrid& ig, const vec& corner1, const vec& corner2, rng& generator, 
				int verbosity, unsigned long long seed, std::vector<std::vector<bias_element> > &bias_batch_list) const {


	/* Definitions from vina1.2 */
	DEBUG_PRINTF("entering CUDA monte_carlo search\n"); //debug

	vec authentic_v(1000, 1000, 1000); // FIXME? this is here to avoid max_fl/max_fl

	quasi_newton quasi_newton_par;
    const int quasi_newton_par_max_steps = local_steps; // no need to decrease step

	/* Allocate CPU memory and define new data structure */
	DEBUG_PRINTF("Allocating CPU memory\n"); //debug
	m_cuda_t *m_cuda;
	checkCUDA(hipHostMalloc(&m_cuda, sizeof(m_cuda_t)));

	output_type_cuda_t *rand_molec_struc_tmp;
	checkCUDA(hipHostMalloc(&rand_molec_struc_tmp, sizeof(output_type_cuda_t)));

	ig_cuda_t *ig_cuda_ptr;
	checkCUDA(hipHostMalloc(&ig_cuda_ptr, sizeof(ig_cuda_t)));

	p_cuda_t_cpu *p_cuda;
	checkCUDA(hipHostMalloc(&p_cuda, sizeof(p_cuda_t_cpu)));


	/* End CPU allocation */

	/* Allocate GPU memory */
	DEBUG_PRINTF("Allocating GPU memory\n");
	size_t m_cuda_size = sizeof(m_cuda_t);
	DEBUG_PRINTF("m_cuda_size=%lu\n", m_cuda_size);
	size_t ig_cuda_size = sizeof(ig_cuda_t);
	DEBUG_PRINTF("ig_cuda_size=%lu\n", ig_cuda_size);
	DEBUG_PRINTF("p_cuda_size_cpu=%lu\n",sizeof(p_cuda_t_cpu));

	size_t p_cuda_size_gpu = sizeof(p_cuda_t);
	DEBUG_PRINTF("p_cuda_size_gpu=%lu\n", p_cuda_size_gpu);

	// rand_molec_struc_gpu
	float *rand_molec_struc_gpu;
	checkCUDA(hipMalloc(&rand_molec_struc_gpu, thread * SIZE_OF_MOLEC_STRUC));
	// best_e_gpu
	float *best_e_gpu;
	float epsilon_fl_float = static_cast<float>(epsilon_fl);
	checkCUDA(hipMalloc(&best_e_gpu, sizeof(float)));
	checkCUDA(hipMemcpy(best_e_gpu, &max_fl, sizeof(float), hipMemcpyHostToDevice));
	
	// use cuRand to generate random values on GPU
	hiprandStatePhilox4_32_10_t* states;
	DEBUG_PRINTF("random states size=%lu\n", sizeof(hiprandStatePhilox4_32_10_t) * thread);
	checkCUDA(hipMalloc(&states, sizeof(hiprandStatePhilox4_32_10_t) * thread));

	// hunt_cap_gpu
	float *hunt_cap_gpu;
	float hunt_cap_float[3] = {static_cast<float>(hunt_cap[0]), static_cast<float>(hunt_cap[1]), static_cast<float>(hunt_cap[2])};

	checkCUDA(hipMalloc(&hunt_cap_gpu, 3 * sizeof(float)));
	// Preparing m related data
	m_cuda_t* m_cuda_gpu;
	DEBUG_PRINTF("m_cuda_size=%lu", m_cuda_size);
	checkCUDA(hipMalloc(&m_cuda_gpu, num_of_ligands * m_cuda_size));
	// Preparing p related data

	p_cuda_t *p_cuda_gpu;
	checkCUDA(hipMalloc(&p_cuda_gpu, num_of_ligands * p_cuda_size_gpu));
	DEBUG_PRINTF("p_cuda_gpu=%p\n", p_cuda_gpu);
	// Preparing ig related data (cache related data)
	ig_cuda_t *ig_cuda_gpu;
	
	float *authentic_v_gpu;
	float authentic_v_float[3] = {static_cast<float>(authentic_v[0]), static_cast<float>(authentic_v[1]),static_cast<float>(authentic_v[2])};

	checkCUDA(hipMalloc(&authentic_v_gpu, sizeof(authentic_v_float)));
	// Preparing result data
	output_type_cuda_t *results_gpu;
	checkCUDA(hipMalloc(&results_gpu, thread * sizeof(output_type_cuda_t)));

	/* End Allocating GPU Memory */


	assert(num_of_ligands <= MAX_LIGAND_NUM);
	assert(thread <= MAX_THREAD);

	struct tmp_struct {
		int start_index = 0;
		int parent_index = 0;
		void store_node(tree<segment>& child_ptr, rigid_cuda_t& rigid) {
			start_index++; // start with index 1, index 0 is root node
			rigid.parent[start_index] = parent_index;
			rigid.atom_range[start_index][0] = child_ptr.node.begin;
			rigid.atom_range[start_index][1] = child_ptr.node.end;
			for (int i = 0; i < 9; i++) rigid.orientation_m[start_index][i] = child_ptr.node.get_orientation_m().data[i];
			rigid.orientation_q[start_index][0] = child_ptr.node.orientation().R_component_1();
			rigid.orientation_q[start_index][1] = child_ptr.node.orientation().R_component_2();
			rigid.orientation_q[start_index][2] = child_ptr.node.orientation().R_component_3();
			rigid.orientation_q[start_index][3] = child_ptr.node.orientation().R_component_4();
			for (int i = 0; i < 3; i++) {
				rigid.origin[start_index][i] = child_ptr.node.get_origin()[i];
				rigid.axis[start_index][i] = child_ptr.node.get_axis()[i];
				rigid.relative_axis[start_index][i] = child_ptr.node.relative_axis[i];
				rigid.relative_origin[start_index][i] = child_ptr.node.relative_origin[i];
			}
			if (child_ptr.children.size() == 0) return;
			else {
				assert(start_index < MAX_NUM_OF_RIGID);
				int parent_index_tmp = start_index;
				for (int i = 0; i < child_ptr.children.size(); i++) {
					this->parent_index = parent_index_tmp; // Update parent index
					this->store_node(child_ptr.children[i], rigid);
				}
			}
		}
	};

	for (int l = 0;l < num_of_ligands; ++l)
	{
		model &m = m_gpu[l];
		const precalculate_byatom &p = p_gpu[l];

		/* Prepare m related data */
		conf_size s = m.get_size();
		change g(s);
		output_type tmp(s, 0);
		tmp.c = m.get_initial_conf();

		assert(m.atoms.size() < MAX_NUM_OF_ATOMS);

		// Preparing ligand data
		DEBUG_PRINTF("prepare ligand data\n");
		assert(m.num_other_pairs() == 0); // m.other_pairs is not supported!
		assert(m.ligands.size() <= 1); // Only one ligand supported!

		if (m.ligands.size() == 0){ // ligand parsing error
			m_cuda->m_num_movable_atoms = -1;
			DEBUG_PRINTF("copy m_cuda to gpu, size=%lu\n", sizeof(m_cuda_t));
			checkCUDA(hipMemcpy(m_cuda_gpu + l, m_cuda, sizeof(m_cuda_t), hipMemcpyHostToDevice));
		}
		else
		{
			for (int i = 0; i < m.atoms.size(); i++) {
				m_cuda->atoms[i].types[0] = m.atoms[i].el;// To store 4 atoms types (el, ad, xs, sy)
				m_cuda->atoms[i].types[1] = m.atoms[i].ad;
				m_cuda->atoms[i].types[2] = m.atoms[i].xs;
				m_cuda->atoms[i].types[3] = m.atoms[i].sy;
				for (int j = 0; j < 3; j++) {
					m_cuda->atoms[i].coords[j] = m.atoms[i].coords[j];// To store atom coords
				}
			}


			// To store atoms coords
			for (int i = 0; i < m.coords.size(); i++) {
				for (int j = 0; j < 3; j++) {
					m_cuda->m_coords.coords[i][j] = m.coords[i].data[j];
				}
			}

			//To store minus forces
			for (int i = 0; i < m.coords.size(); i++) {
				for (int j = 0; j < 3; j++) {
					m_cuda->minus_forces.coords[i][j] = m.minus_forces[i].data[j];
				}
			}

			m_cuda->ligand.pairs.num_pairs = m.ligands[0].pairs.size();
			for (int i = 0; i < m_cuda->ligand.pairs.num_pairs; i++) {
				m_cuda->ligand.pairs.type_pair_index[i]	= m.ligands[0].pairs[i].type_pair_index;
				m_cuda->ligand.pairs.a[i]					= m.ligands[0].pairs[i].a;
				m_cuda->ligand.pairs.b[i]					= m.ligands[0].pairs[i].b;
			}
			m_cuda->ligand.begin = m.ligands[0].begin; // 0
			m_cuda->ligand.end = m.ligands[0].end; // 29
			ligand &m_ligand = m.ligands[0]; // Only support one ligand
			DEBUG_PRINTF("m_ligand.end=%lu, MAX_NUM_OF_ATOMS=%d\n", m_ligand.end, MAX_NUM_OF_ATOMS);
			assert(m_ligand.end < MAX_NUM_OF_ATOMS);

			// Store root node
			m_cuda->ligand.rigid.atom_range[0][0] = m_ligand.node.begin;
			m_cuda->ligand.rigid.atom_range[0][1] = m_ligand.node.end;
			for (int i = 0; i < 3; i++) m_cuda->ligand.rigid.origin[0][i] = m_ligand.node.get_origin()[i];
			for (int i = 0; i < 9; i++) m_cuda->ligand.rigid.orientation_m[0][i] = m_ligand.node.get_orientation_m().data[i];
			m_cuda->ligand.rigid.orientation_q[0][0] = m_ligand.node.orientation().R_component_1();
			m_cuda->ligand.rigid.orientation_q[0][1] = m_ligand.node.orientation().R_component_2();
			m_cuda->ligand.rigid.orientation_q[0][2] = m_ligand.node.orientation().R_component_3();
			m_cuda->ligand.rigid.orientation_q[0][3] = m_ligand.node.orientation().R_component_4();
			for (int i = 0; i < 3; i++) {m_cuda->ligand.rigid.axis[0][i] = 0;m_cuda->ligand.rigid.relative_axis[0][i] = 0;m_cuda->ligand.rigid.relative_origin[0][i] = 0;}

			// Store children nodes (in depth-first order)
			DEBUG_PRINTF("store children nodes\n");

			tmp_struct ts;
			for (int i = 0; i < m_ligand.children.size(); i++) {
				ts.parent_index = 0; // Start a new branch, whose parent is 0
				ts.store_node(m_ligand.children[i], m_cuda->ligand.rigid);
			}
			m_cuda->ligand.rigid.num_children = ts.start_index;

			// set children_map
			DEBUG_PRINTF("set children map\n");
			for (int i = 0; i < MAX_NUM_OF_RIGID; i++)
				for (int j = 0; j < MAX_NUM_OF_RIGID; j++)
					m_cuda->ligand.rigid.children_map[i][j] = false;
			for (int i = 1; i < m_cuda->ligand.rigid.num_children + 1; i++) {
				int parent_index = m_cuda->ligand.rigid.parent[i];
				m_cuda->ligand.rigid.children_map[parent_index][i] = true;
			}
			m_cuda->m_num_movable_atoms = m.num_movable_atoms();

			DEBUG_PRINTF("copy m_cuda to gpu, size=%lu\n", sizeof(m_cuda_t));
			checkCUDA(hipMemcpy(m_cuda_gpu + l, m_cuda, sizeof(m_cuda_t), hipMemcpyHostToDevice));

			/* Prepare rand_molec_struc data */
			int lig_torsion_size = tmp.c.ligands[0].torsions.size();
			DEBUG_PRINTF("lig_torsion_size=%d\n", lig_torsion_size);
			int flex_torsion_size;
			if (tmp.c.flex.size() != 0) flex_torsion_size = tmp.c.flex[0].torsions.size();
			else flex_torsion_size = 0;
			// std::vector<vec> uniform_data;
			// uniform_data.resize(thread);

			for (int i = 0; i < threads_per_ligand; ++i){
				if (!local_only){
					tmp.c.randomize(corner1, corner2, generator); // generate a random structure, can move to GPU if necessary
				}
				for (int j = 0; j < 3; j++) rand_molec_struc_tmp->position[j] = tmp.c.ligands[0].rigid.position[j];
				assert(lig_torsion_size <= MAX_NUM_OF_LIG_TORSION);
				for (int j = 0; j < lig_torsion_size; j++) rand_molec_struc_tmp->lig_torsion[j] = tmp.c.ligands[0].torsions[j];// Only support one ligand
				assert(flex_torsion_size <= MAX_NUM_OF_FLEX_TORSION);
				for (int j = 0; j < flex_torsion_size; j++) rand_molec_struc_tmp->flex_torsion[j] = tmp.c.flex[0].torsions[j];// Only support one flex

				rand_molec_struc_tmp->orientation[0] = (float)tmp.c.ligands[0].rigid.orientation.R_component_1();
				rand_molec_struc_tmp->orientation[1] = (float)tmp.c.ligands[0].rigid.orientation.R_component_2();
				rand_molec_struc_tmp->orientation[2] = (float)tmp.c.ligands[0].rigid.orientation.R_component_3();
				rand_molec_struc_tmp->orientation[3] = (float)tmp.c.ligands[0].rigid.orientation.R_component_4();

				rand_molec_struc_tmp->lig_torsion_size = lig_torsion_size;

				float *rand_molec_struc_gpu_tmp = rand_molec_struc_gpu + (l * threads_per_ligand + i) * SIZE_OF_MOLEC_STRUC / sizeof(float);
				checkCUDA(hipMemcpy(rand_molec_struc_gpu_tmp, rand_molec_struc_tmp, SIZE_OF_MOLEC_STRUC, hipMemcpyHostToDevice));

			}

			/* Preparing p related data */
			DEBUG_PRINTF("Preaparing p related data\n"); //debug

			// copy pointer instead of data
			p_cuda->m_cutoff_sqr = p.m_cutoff_sqr;
			p_cuda->factor = p.m_factor;
			p_cuda->n = p.m_n;
			p_cuda->m_data_size = p.m_data.m_data.size();
			checkCUDA(hipMemcpy(p_cuda_gpu + l, p_cuda, sizeof(p_cuda_t), hipMemcpyHostToDevice));
			checkCUDA(hipMemcpy(&(p_cuda_gpu[l].m_data), &(m_data_list_gpu[l].p_data), sizeof(p_m_data_cuda_t *), hipMemcpyHostToDevice)); // check if fl == float

		}

	}

	/* Prepare data only concerns rigid receptor */

    // Preparing igrid related data
	DEBUG_PRINTF("Preparing ig related data\n"); //debug
	
	bool multi_bias = (bias_batch_list.size() == num_of_ligands);
	if (multi_bias){
		// multi bias mode
		std::cout << "with multi bias ";
		
		checkCUDA(hipMalloc(&ig_cuda_gpu, ig_cuda_size * num_of_ligands));
		for (int l = 0;l < num_of_ligands;++l){
			if (ig.get_atu() == atom_type::XS)
			{
				cache ig_tmp(ig.get_gd(), ig.get_slope());
				ig_tmp.m_grids = ig.get_grids();
				// // debug
				// if (l == 1){
				// 	std::cout << "writing original grid map\n";
				// 	ig_tmp.write(std::string("./ori"), szv(1,0));
				// }
				ig_tmp.compute_bias(m_gpu[l], bias_batch_list[l]);
				// // debug
				// std::cout << "writing bias\n";
				// ig_tmp.write(std::string("./")+std::to_string(l), szv(1,0));
				ig_cuda_ptr->atu = ig.get_atu(); // atu
				DEBUG_PRINTF("ig_cuda_ptr->atu=%d\n", ig_cuda_ptr->atu);
				ig_cuda_ptr->slope = ig.get_slope(); // slope
				std::vector<grid> tmp_grids = ig.get_grids();
				int grid_size = tmp_grids.size();
				DEBUG_PRINTF("ig.size()=%d, GRIDS_SIZE=%d, should be 33\n", grid_size, GRIDS_SIZE);

				for (int i = 0; i < grid_size; i++) {
					// DEBUG_PRINTF("i=%d\n",i); //debug
					for (int j = 0; j < 3; j++) {
						ig_cuda_ptr->grids[i].m_init[j] = tmp_grids[i].m_init[j];
						ig_cuda_ptr->grids[i].m_factor[j] = tmp_grids[i].m_factor[j];
						ig_cuda_ptr->grids[i].m_dim_fl_minus_1[j] = tmp_grids[i].m_dim_fl_minus_1[j];
						ig_cuda_ptr->grids[i].m_factor_inv[j] = tmp_grids[i].m_factor_inv[j];
					}
					if (tmp_grids[i].m_data.dim0() != 0) {
						ig_cuda_ptr->grids[i].m_i = tmp_grids[i].m_data.dim0(); assert(MAX_NUM_OF_GRID_MI >= ig_cuda_ptr->grids[i].m_i);
						ig_cuda_ptr->grids[i].m_j = tmp_grids[i].m_data.dim1(); assert(MAX_NUM_OF_GRID_MJ >= ig_cuda_ptr->grids[i].m_j);
						ig_cuda_ptr->grids[i].m_k = tmp_grids[i].m_data.dim2(); assert(MAX_NUM_OF_GRID_MK >= ig_cuda_ptr->grids[i].m_k);

						assert(tmp_grids[i].m_data.m_data.size()==ig_cuda_ptr->grids[i].m_i * ig_cuda_ptr->grids[i].m_j * ig_cuda_ptr->grids[i].m_k);
						assert(tmp_grids[i].m_data.m_data.size() <= MAX_NUM_OF_GRID_POINT);
						memcpy(ig_cuda_ptr->grids[i].m_data, tmp_grids[i].m_data.m_data.data(), tmp_grids[i].m_data.m_data.size() * sizeof(fl));
					}
					else {
						ig_cuda_ptr->grids[i].m_i = 0;
						ig_cuda_ptr->grids[i].m_j = 0;
						ig_cuda_ptr->grids[i].m_k = 0;
					}
				}
			}
			else{
				ad4cache ig_tmp(ig.get_slope());
				ig_tmp.m_grids = ig.get_grids();
				// // debug
				// if (l == 1){
				// 	std::cout << "writing original grid map\n";
				// 	ig_tmp.write(std::string("./ori"), szv(1,0));
				// }
				ig_tmp.set_bias(bias_batch_list[l]);
				// // debug
				// std::cout << "writing bias\n";
				// ig_tmp.write(std::string("./")+std::to_string(l), szv(1,0));
				ig_cuda_ptr->atu = ig.get_atu(); // atu
				DEBUG_PRINTF("ig_cuda_ptr->atu=%d\n", ig_cuda_ptr->atu);
				ig_cuda_ptr->slope = ig.get_slope(); // slope
				std::vector<grid> tmp_grids = ig.get_grids();
				int grid_size = tmp_grids.size();
				DEBUG_PRINTF("ig.size()=%d, GRIDS_SIZE=%d, should be 33\n", grid_size, GRIDS_SIZE);

				for (int i = 0; i < grid_size; i++) {
					// DEBUG_PRINTF("i=%d\n",i); //debug
					for (int j = 0; j < 3; j++) {
						ig_cuda_ptr->grids[i].m_init[j] = tmp_grids[i].m_init[j];
						ig_cuda_ptr->grids[i].m_factor[j] = tmp_grids[i].m_factor[j];
						ig_cuda_ptr->grids[i].m_dim_fl_minus_1[j] = tmp_grids[i].m_dim_fl_minus_1[j];
						ig_cuda_ptr->grids[i].m_factor_inv[j] = tmp_grids[i].m_factor_inv[j];
					}
					if (tmp_grids[i].m_data.dim0() != 0) {
						ig_cuda_ptr->grids[i].m_i = tmp_grids[i].m_data.dim0(); assert(MAX_NUM_OF_GRID_MI >= ig_cuda_ptr->grids[i].m_i);
						ig_cuda_ptr->grids[i].m_j = tmp_grids[i].m_data.dim1(); assert(MAX_NUM_OF_GRID_MJ >= ig_cuda_ptr->grids[i].m_j);
						ig_cuda_ptr->grids[i].m_k = tmp_grids[i].m_data.dim2(); assert(MAX_NUM_OF_GRID_MK >= ig_cuda_ptr->grids[i].m_k);

						assert(tmp_grids[i].m_data.m_data.size()==ig_cuda_ptr->grids[i].m_i * ig_cuda_ptr->grids[i].m_j * ig_cuda_ptr->grids[i].m_k);
						memcpy(ig_cuda_ptr->grids[i].m_data, tmp_grids[i].m_data.m_data.data(), tmp_grids[i].m_data.m_data.size() * sizeof(fl));
					}
					else {
						ig_cuda_ptr->grids[i].m_i = 0;
						ig_cuda_ptr->grids[i].m_j = 0;
						ig_cuda_ptr->grids[i].m_k = 0;
					}
				}
			}
			
			checkCUDA(hipMemcpy(ig_cuda_gpu+l, ig_cuda_ptr, ig_cuda_size, hipMemcpyHostToDevice));
		
		}
		std::cout << "set\n";
	}
	else{

		ig_cuda_ptr->atu = ig.get_atu(); // atu
		DEBUG_PRINTF("ig_cuda_ptr->atu=%d\n", ig_cuda_ptr->atu);
		ig_cuda_ptr->slope = ig.get_slope(); // slope
		std::vector<grid> tmp_grids = ig.get_grids();
		int grid_size = tmp_grids.size();
		DEBUG_PRINTF("ig.size()=%d, GRIDS_SIZE=%d, should be 33\n", grid_size, GRIDS_SIZE);

		for (int i = 0; i < grid_size; i++) {
			// DEBUG_PRINTF("i=%d\n",i); //debug
			for (int j = 0; j < 3; j++) {
				ig_cuda_ptr->grids[i].m_init[j] = tmp_grids[i].m_init[j];
				ig_cuda_ptr->grids[i].m_factor[j] = tmp_grids[i].m_factor[j];
				ig_cuda_ptr->grids[i].m_dim_fl_minus_1[j] = tmp_grids[i].m_dim_fl_minus_1[j];
				ig_cuda_ptr->grids[i].m_factor_inv[j] = tmp_grids[i].m_factor_inv[j];
			}
			if (tmp_grids[i].m_data.dim0() != 0) {
				ig_cuda_ptr->grids[i].m_i = tmp_grids[i].m_data.dim0(); assert(MAX_NUM_OF_GRID_MI >= ig_cuda_ptr->grids[i].m_i);
				ig_cuda_ptr->grids[i].m_j = tmp_grids[i].m_data.dim1(); assert(MAX_NUM_OF_GRID_MJ >= ig_cuda_ptr->grids[i].m_j);
				ig_cuda_ptr->grids[i].m_k = tmp_grids[i].m_data.dim2(); assert(MAX_NUM_OF_GRID_MK >= ig_cuda_ptr->grids[i].m_k);

				assert(tmp_grids[i].m_data.m_data.size()==ig_cuda_ptr->grids[i].m_i * ig_cuda_ptr->grids[i].m_j * ig_cuda_ptr->grids[i].m_k);
				memcpy(ig_cuda_ptr->grids[i].m_data, tmp_grids[i].m_data.m_data.data(), tmp_grids[i].m_data.m_data.size() * sizeof(fl));
			}
			else {
				ig_cuda_ptr->grids[i].m_i = 0;
				ig_cuda_ptr->grids[i].m_j = 0;
				ig_cuda_ptr->grids[i].m_k = 0;
			}
		}
		DEBUG_PRINTF("memcpy ig_cuda, ig_cuda_size=%lu\n", ig_cuda_size);
		checkCUDA(hipMalloc(&ig_cuda_gpu, ig_cuda_size));
		checkCUDA(hipMemcpy(ig_cuda_gpu, ig_cuda_ptr, ig_cuda_size, hipMemcpyHostToDevice));
		
	}


	float mutation_amplitude_float = static_cast<float>(mutation_amplitude);

	checkCUDA(hipMemcpy(hunt_cap_gpu, hunt_cap_float, 3 * sizeof(float), hipMemcpyHostToDevice));
	float hunt_test[3];
	checkCUDA(hipMemcpy(hunt_test, hunt_cap_gpu, 3 * sizeof(float), hipMemcpyDeviceToHost));
	DEBUG_PRINTF("hunt_test[1]=%f, hunt_cap_float[1]=%f\n", hunt_test[1], hunt_cap_float[1]);
	checkCUDA(hipMemcpy(authentic_v_gpu, authentic_v_float, sizeof(authentic_v_float), hipMemcpyHostToDevice));



	/* Add timing */
	hipEvent_t start,stop;
	checkCUDA(hipEventCreate(&start));
	checkCUDA(hipEventCreate(&stop));
	checkCUDA(hipEventRecord(start,NULL));

	/* Launch kernel */
	DEBUG_PRINTF("launch kernel, global_steps=%d, thread=%d, num_of_ligands=%d\n", global_steps, thread, num_of_ligands);
	kernel<<<thread / 32 + 1, 32>>>(m_cuda_gpu, ig_cuda_gpu, p_cuda_gpu, rand_molec_struc_gpu,
		best_e_gpu, quasi_newton_par_max_steps, mutation_amplitude_float,
		states, seed,
		epsilon_fl_float, hunt_cap_gpu, authentic_v_gpu, results_gpu, global_steps,
		num_of_ligands, threads_per_ligand, multi_bias);

	// Device to Host memcpy of precalculated_byatom, copy back data to p_gpu
    p_m_data_cuda_t *p_data;
    checkCUDA(hipHostMalloc(&p_data, sizeof(p_m_data_cuda_t) * MAX_P_DATA_M_DATA_SIZE));
	output_type_cuda_t *results;
	checkCUDA(hipHostMalloc(&results, thread * sizeof(output_type_cuda_t)));

	for (int l = 0;l < num_of_ligands; ++l){
        // copy data to m_data on CPU, then to p_gpu[l]
		int pnum = p_gpu[l].m_data.m_data.size();
        checkCUDA(hipMemcpy(p_data, m_data_list_gpu[l].p_data, sizeof(p_m_data_cuda_t) * pnum, hipMemcpyDeviceToHost));
        checkCUDA(hipFree(m_data_list_gpu[l].p_data)); // free m_cuda pointers in p_cuda
        for (int i = 0;i < pnum; ++i){
            memcpy(&p_gpu[l].m_data.m_data[i].fast[0], p_data[i].fast, sizeof(p_data[i].fast));
            memcpy(&p_gpu[l].m_data.m_data[i].smooth[0], p_data[i].smooth, sizeof(p_data[i].smooth));
        }
    }
	// DEBUG_PRINTF("energies about the first ligand on GPU:\n");
    // for (int i = 0;i < 20; ++i){
    //     DEBUG_PRINTF("precalculated_byatom.m_data.m_data[%d]: (smooth.first, smooth.second, fast) ", i);
    //     for (int j = 0;j < FAST_SIZE; ++j){
    //         DEBUG_PRINTF("(%f, %f, %f) ", p_gpu[0].m_data.m_data[i].smooth[j].first,
    //         p_gpu[0].m_data.m_data[i].smooth[j].second, p_gpu[0].m_data.m_data[i].fast[j]);
    //     }
    //     DEBUG_PRINTF("\n");
    // }

	checkCUDA(hipDeviceSynchronize());
	/* Timing output */

	checkCUDA(hipEventRecord(stop,NULL));
	hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);
	DEBUG_PRINTF("Time spend on GPU is %f ms\n", msecTotal);

	/* Convert result data. Can be improved by mapping memory
	*/
	DEBUG_PRINTF("cuda to vina\n");

	checkCUDA(hipMemcpy(results, results_gpu, thread * sizeof(output_type_cuda_t), hipMemcpyDeviceToHost));

	std::vector<output_type> result_vina = cuda_to_vina(results, thread);

	DEBUG_PRINTF("result size=%lu\n", result_vina.size());


	for (int i = 0; i < thread; ++i){
		add_to_output_container(out_gpu[i / threads_per_ligand], result_vina[i], min_rmsd, num_saved_mins);
	}
	for (int i = 0; i < num_of_ligands; ++i){
		DEBUG_PRINTF("output poses size = %lu\n", out_gpu[i].size());
		if (out_gpu[i].size() == 0) continue;
		DEBUG_PRINTF("output poses energy from gpu =");
		for (int j = 0;j < out_gpu[i].size(); ++j)
			DEBUG_PRINTF("%f ", out_gpu[i][j].e);
		DEBUG_PRINTF("\n");
	}

	/* Free memory */
	checkCUDA(hipFree(m_cuda_gpu));
	checkCUDA(hipFree(ig_cuda_gpu));
	checkCUDA(hipFree(p_cuda_gpu));
	checkCUDA(hipFree(rand_molec_struc_gpu));
	checkCUDA(hipFree(best_e_gpu));
	checkCUDA(hipFree(hunt_cap_gpu));
	checkCUDA(hipFree(authentic_v_gpu));
	checkCUDA(hipFree(results_gpu));
	checkCUDA(hipFree(states));
	checkCUDA(hipHostFree(m_cuda));
	checkCUDA(hipHostFree(rand_molec_struc_tmp));
	checkCUDA(hipHostFree(ig_cuda_ptr));
	checkCUDA(hipHostFree(p_cuda));
	checkCUDA(hipHostFree(p_data));
	checkCUDA(hipHostFree(results));

	DEBUG_PRINTF("exit monte_carlo\n");

}

bool metropolis_accept(fl old_f, fl new_f, fl temperature, rng& generator) {
	if(new_f < old_f) return true;
	const fl acceptance_probability = std::exp((old_f - new_f) / temperature);
	return random_fl(0, 1, generator) < acceptance_probability;
}

__host__
void monte_carlo::operator()(model& m, output_container& out, const precalculate_byatom& p, const igrid& ig, const vec& corner1, const vec& corner2, rng& generator) const {
    int evalcount = 0;
	vec authentic_v(1000, 1000, 1000); // FIXME? this is here to avoid max_fl/max_fl
	conf_size s = m.get_size();
	change g(s);
	output_type tmp(s, 0);
	tmp.c.randomize(corner1, corner2, generator);
	fl best_e = max_fl;
	quasi_newton quasi_newton_par;
    quasi_newton_par.max_steps = local_steps;
	VINA_U_FOR(step, global_steps) {
		// if(increment_me)
		// 	++(*increment_me);
		if((max_evals > 0) & (evalcount > max_evals))
			break;
		output_type candidate = tmp;
		mutate_conf(candidate.c, m, mutation_amplitude, generator);
		quasi_newton_par(m, p, ig, candidate, g, hunt_cap, evalcount);
		if(step == 0 || metropolis_accept(tmp.e, candidate.e, temperature, generator)) {
			tmp = candidate;

			m.set(tmp.c); // FIXME? useless?

			// FIXME only for very promising ones
			if(tmp.e < best_e || out.size() < num_saved_mins) {
				quasi_newton_par(m, p, ig, tmp, g, authentic_v, evalcount);
				m.set(tmp.c); // FIXME? useless?
				tmp.coords = m.get_heavy_atom_movable_coords();
				add_to_output_container(out, tmp, min_rmsd, num_saved_mins); // 20 - max size
				if(tmp.e < best_e)
					best_e = tmp.e;
			}
		}
	}
	VINA_CHECK(!out.empty());
	VINA_CHECK(out.front().e <= out.back().e); // make sure the sorting worked in the correct order
}


/* Above based on monte-carlo.cpp */

// #endif
